#include "cuda_utils.cuh"
#include <hipblas.h>
#include <iostream>
#include <vector>
#define ALIGN 32

int main(int argc, char *argv[])
{
    auto args = getTestArgs(argc, argv);
    auto M = args.at("M");
    auto N = args.at("N");
    auto K = args.at("K");
    auto iters = args.at("iters");

    // initialize host arrays
    cf_t *A = (cf_t *)aligned_alloc(ALIGN, sizeof(cf_t) * M * K);
    cf_t *B = (cf_t *)aligned_alloc(ALIGN, sizeof(cf_t) * K * N);
    cf_t *C = (cf_t *)aligned_alloc(ALIGN, sizeof(cf_t) * M * N);
    hipComplex *h_A, *h_B, *h_C;
    hipHostAlloc((void **)&h_A, sizeof(hipComplex) * M * K, hipHostMallocDefault);
    hipHostAlloc((void **)&h_B, sizeof(hipComplex) * K * N, hipHostMallocDefault);
    hipHostAlloc((void **)&h_C, sizeof(hipComplex) * M * N, hipHostMallocDefault);
    cf_t alpha, beta;
    hipComplex d_alpha, d_beta;

    // initialize device arrays
    hipComplex *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, sizeof(hipComplex) * M * K);
    hipMalloc((void **)&d_B, sizeof(hipComplex) * K * N);
    hipMalloc((void **)&d_C, sizeof(hipComplex) * M * N);

    // initialize results
    std::vector<double> ms_results, memcpy_d2h_results, memcpy_h2d_results;
    hipEvent_t start, stop;

    // initialize cudaEvent
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // initialize cuda, cublas handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // warm-up run
    float warmup;
    setArrays((cf_t *)A, B, C, &alpha, &beta, M, N, K);
    memcpyPinned(h_A, h_B, h_C, &d_alpha, &d_beta, A, B, C, &alpha, &beta, M, N, K);
    Arrays2Device(d_A, d_B, d_C, h_A, h_B, h_C, M, N, K);
    hipEventRecord(start);
    hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &d_alpha, d_B, N, d_A, K, &d_beta, d_C, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&warmup, start, stop);
    Array2Host(h_C, d_C, M, N, K);

    for (int i = 0; i < iters; i++)
    {
        setArrays((cf_t *)A, B, C, &alpha, &beta, M, N, K);
        memcpyPinned(h_A, h_B, h_C, &d_alpha, &d_beta, A, B, C, &alpha, &beta, M, N, K);
        auto mem_h2d_result = Arrays2Device(d_A, d_B, d_C, h_A, h_B, h_C, M, N, K);
        memcpy_h2d_results.push_back(mem_h2d_result.second);
        hipEventRecord(start);
        hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &d_alpha, d_B, N, d_A, K, &d_beta, d_C, N);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        auto mem_d2h_result = Array2Host(h_C, d_C, M, N, K);
        memcpy_d2h_results.push_back(mem_d2h_result.second);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        ms_results.push_back(milliseconds);
    }
    printResults(ms_results, memcpy_h2d_results, memcpy_d2h_results);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);
    free(A);
    free(B);
    free(C);
    hipblasDestroy(handle);
    return 0;
}