#include "hip/hip_runtime.h"
#include "cuda_utils.cuh"
#include <hipblas.h>
#include <iostream>
#include <vector>
#define ALIGN 32

int main_part_single_stream(int M, int N, int K, int iters)
{
    // initialize host arrays
    cf_t *A = (cf_t *)aligned_alloc(ALIGN, sizeof(cf_t) * M * K);
    cf_t *B = (cf_t *)aligned_alloc(ALIGN, sizeof(cf_t) * K * N);
    cf_t *C = (cf_t *)aligned_alloc(ALIGN, sizeof(cf_t) * M * N);
    hipComplex *h_A, *h_B, *h_C;
    hipHostAlloc((void **)&h_A, sizeof(hipComplex) * M * K, hipHostMallocDefault);
    hipHostAlloc((void **)&h_B, sizeof(hipComplex) * K * N, hipHostMallocDefault);
    hipHostAlloc((void **)&h_C, sizeof(hipComplex) * M * N, hipHostMallocDefault);
    cf_t alpha, beta;
    hipComplex d_alpha, d_beta;

    // initialize device arrays
    hipComplex *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, sizeof(hipComplex) * M * K);
    hipMalloc((void **)&d_B, sizeof(hipComplex) * K * N);
    hipMalloc((void **)&d_C, sizeof(hipComplex) * M * N);

    // initialize results
    std::vector<double> ms_results, memcpy_d2h_results, memcpy_h2d_results;
    hipEvent_t start, stop;

    // initialize cudaEvent
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // initialize cuda, cublas handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // warm-up run
    float warmup;
    setArrays((cf_t *)A, B, C, &alpha, &beta, M, N, K);
    memcpyPinned(h_A, h_B, h_C, &d_alpha, &d_beta, A, B, C, &alpha, &beta, M, N, K);
    Arrays2Device(d_A, d_B, d_C, h_A, h_B, h_C, M, N, K);
    hipEventRecord(start);
    hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &d_alpha, d_B, N, d_A, K, &d_beta, d_C, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&warmup, start, stop);
    Array2Host(h_C, d_C, M, N, K);

    for (int i = 0; i < iters; i++)
    {
        setArrays((cf_t *)A, B, C, &alpha, &beta, M, N, K);
        memcpyPinned(h_A, h_B, h_C, &d_alpha, &d_beta, A, B, C, &alpha, &beta, M, N, K);
        auto mem_h2d_result = Arrays2Device(d_A, d_B, d_C, h_A, h_B, h_C, M, N, K);
        memcpy_h2d_results.push_back(mem_h2d_result.second);
        hipEventRecord(start);
        hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &d_alpha, d_B, N, d_A, K, &d_beta, d_C, N);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        auto mem_d2h_result = Array2Host(h_C, d_C, M, N, K);
        memcpy_d2h_results.push_back(mem_d2h_result.second);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        ms_results.push_back(milliseconds);
    }
    printResults(ms_results, memcpy_h2d_results, memcpy_d2h_results);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);
    free(A);
    free(B);
    free(C);
    hipblasDestroy(handle);
    cudaErrorHandle(hipDeviceReset());
    return 0;
}

int main_part_multiple_stream(int M, int N, int K, int divisions, int iters)
{
    int n = ceil((double)N/divisions);

    // initialize host arrays
    cf_t *A = (cf_t *)aligned_alloc(ALIGN, sizeof(cf_t) * M * K);
    cf_t *B = (cf_t *)aligned_alloc(ALIGN, sizeof(cf_t) * K * N);
    cf_t *C = (cf_t *)aligned_alloc(ALIGN, sizeof(cf_t) * M * N);
    hipComplex *h_A, *h_B, *h_C;
    hipHostAlloc((void **)&h_A, sizeof(hipComplex) * M * K, hipHostMallocDefault);
    hipHostAlloc((void **)&h_B, sizeof(hipComplex) * K * N, hipHostMallocDefault);
    hipHostAlloc((void **)&h_C, sizeof(hipComplex) * M * N, hipHostMallocDefault);
    cf_t alpha, beta;
    hipComplex d_alpha, d_beta;

    // initialize device arrays
    hipComplex *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, sizeof(hipComplex) * M * K);
    hipMalloc((void **)&d_B, sizeof(hipComplex) * K * N);
    hipMalloc((void **)&d_C, sizeof(hipComplex) * M * N);

    // initialize results
    std::vector<double> ms_results, memcpy_d2h_results, memcpy_h2d_results;
    hipEvent_t start, stop;

    // initialize cudaEvent
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // initialize cudaHandles and cudaStreams
    std::vector<cudaStreamHandle_t> cudaStreamHandles(divisions);
    // std::vector<hipblasHandle_t> cudaHandles(divisions);
    // std::vector<hipStream_t> cudaStreams(divisions);
    for (int k = 0; k < divisions; k++)
    {
        hipblasCreate(&cudaStreamHandles[k].first);
        hipStreamCreate(&cudaStreamHandles[k].second);
        hipblasSetStream(cudaStreamHandles[k].first, cudaStreamHandles[k].second);
    }

    // 
    for (int i = 0; i < iters+1; i++)
    {
        setArrays((cf_t *)A, B, C, &alpha, &beta, M, N, K);
        memcpyPinned(h_A, h_B, h_C, &d_alpha, &d_beta, A, B, C, &alpha, &beta, M, N, K);
        auto mem_h2d_result = Arrays2DeviceWithStreams(d_A, d_B, d_C, h_A, h_B, h_C, M, N, K, divisions, cudaStreamHandles);
        hipEventRecord(start);
        for (int k = 0; k < divisions; k++)
        {
            auto start_point = k*n;
            auto end_point = start_point + n;
            auto _n = end_point < N ? n : N-start_point;
            hipblasCgemm(
                cudaStreamHandles[k].first,
                HIPBLAS_OP_N,
                HIPBLAS_OP_N,
                _n,
                M,
                K,
                &d_alpha,
                &d_B[start_point],
                _n,
                d_A,
                K,
                &d_beta,
                &d_C[start_point],
                _n
            );
        }
        hipEventRecord(stop);

        auto mem_d2h_result = Arrays2HostWithStreams(h_C, d_C, M, N, K, divisions, cudaStreamHandles);

        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        
        // 記録
        if (i != 0)
        {
            ms_results.push_back(milliseconds);
            memcpy_h2d_results.push_back(mem_h2d_result.second);
            memcpy_d2h_results.push_back(mem_d2h_result.second);
        }
    }
    printResults(ms_results, memcpy_h2d_results, memcpy_d2h_results);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);
    free(A);
    free(B);
    free(C);
    for (int k = 0; k < divisions; k++)
    {
        hipblasDestroy(cudaStreamHandles[k].first);
        hipStreamDestroy(cudaStreamHandles[k].second);
    }
    cudaErrorHandle(hipDeviceReset());
    return 0;
}

int main(int argc, char *argv[])
{
    auto args = getTestArgs(argc, argv);
    auto M = args.at("M");
    auto N = args.at("N");
    auto K = args.at("K");
    auto divisions = args.at("divisions");
    auto iters = args.at("iters");
    
    if (divisions > 0)
    {
        return main_part_multiple_stream(M, N, K, divisions, iters);
    }
    else
    {
        return main_part_single_stream(M, N, K, iters);
    }
}