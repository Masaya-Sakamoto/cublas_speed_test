#include "cuda_utils.cuh"
#include <hipblas.h>
#include <iostream>
#include <vector>
#define ALIGN 32

int main(int argc, char *argv[])
{
    auto args = getTestArgs(argc, argv);
    auto M = args.at("M");
    auto N = args.at("N");
    auto K = args.at("K");
    auto divisions = args.at("divisions");
    auto iters = args.at("iters");

    // initialize host arrays
    cf_t *A = (cf_t *)aligned_alloc(ALIGN, sizeof(cf_t) * M * K);
    cf_t *B = (cf_t *)aligned_alloc(ALIGN, sizeof(cf_t) * K * N);
    cf_t *C = (cf_t *)aligned_alloc(ALIGN, sizeof(cf_t) * M * N);
    hipComplex *h_A, *h_B, *h_C;
    hipHostAlloc((void **)&h_A, sizeof(hipComplex) * M * K, hipHostMallocDefault);
    hipHostAlloc((void **)&h_B, sizeof(hipComplex) * K * N, hipHostMallocDefault);
    hipHostAlloc((void **)&h_C, sizeof(hipComplex) * M * N, hipHostMallocDefault);
    cf_t alpha, beta;
    hipComplex d_alpha, d_beta;

    // initialize device arrays
    hipComplex *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, sizeof(hipComplex) * M * K);
    hipMalloc((void **)&d_B, sizeof(hipComplex) * K * N);
    hipMalloc((void **)&d_C, sizeof(hipComplex) * M * N);

    // initialize results
    std::vector<double> ms_results, memcpy_d2h_results, memcpy_h2d_results;
    hipEvent_t start, stop;

    // initialize cudaEvent
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // initialize cudaHandles and cudaStreams
    std::vector<hipblasHandle_t> cudaHandles(divisions);
    std::vector<hipStream_t> cudaStreams(divisions);
    for (int k = 0; k < divisions; k++)
    {
        hipblasCreate(&cudaHandles[k]);
        hipStreamCreate(&cudaStreams[k]);
        hipblasSetStream(cudaHandles[k], cudaStreams[k]); // Associate cuBLAS with stream
    }

    // 
    for (int i = 0; i < iters; i++)
    {
        setArrays((cf_t *)A, B, C, &alpha, &beta, M, N, K);
        memcpyPinned(h_A, h_B, h_C, &d_alpha, &d_beta, A, B, C, &alpha, &beta, M, N, K);
        
    }
}