#include "hip/hip_runtime.h"
#include "cuda_utils.cuh"
#include <hipblas.h>
#include <iostream>
#include <vector>
#define ALIGN 32

int main(int argc, char *argv[])
{
    auto args = getTestArgs(argc, argv);
    auto M = args.at("M");
    auto N = args.at("N");
    auto K = args.at("K");
    auto divisions = args.at("divisions");
    auto iters = args.at("iters");
    int n = ceil((double)N/divisions);

    // initialize host arrays
    cf_t *A = (cf_t *)aligned_alloc(ALIGN, sizeof(cf_t) * M * K);
    cf_t *B = (cf_t *)aligned_alloc(ALIGN, sizeof(cf_t) * K * N);
    cf_t *C = (cf_t *)aligned_alloc(ALIGN, sizeof(cf_t) * M * N);
    hipComplex *h_A, *h_B, *h_C;
    hipHostAlloc((void **)&h_A, sizeof(hipComplex) * M * K, hipHostMallocDefault);
    hipHostAlloc((void **)&h_B, sizeof(hipComplex) * K * N, hipHostMallocDefault);
    hipHostAlloc((void **)&h_C, sizeof(hipComplex) * M * N, hipHostMallocDefault);
    cf_t alpha, beta;
    hipComplex d_alpha, d_beta;

    // initialize device arrays
    hipComplex *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, sizeof(hipComplex) * M * K);
    hipMalloc((void **)&d_B, sizeof(hipComplex) * K * N);
    hipMalloc((void **)&d_C, sizeof(hipComplex) * M * N);

    // initialize results
    std::vector<double> memcpy_d2h_results, memcpy_h2d_results;
    std::vector<double> ms_results{-1};
    hipEvent_t start, stop;

    // initialize cudaEvent
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // initialize cudaHandles and cudaStreams
    std::vector<cudaStreamHandle_t> cudaStreamHandles(divisions);
    // std::vector<hipblasHandle_t> cudaHandles(divisions);
    // std::vector<hipStream_t> cudaStreams(divisions);
    for (int k = 0; k < divisions; k++)
    {
        hipblasCreate(&cudaStreamHandles[k].first);
        hipStreamCreate(&cudaStreamHandles[k].second);
        hipblasSetStream(cudaStreamHandles[k].first, cudaStreamHandles[k].second);
    }

    // 
    for (int i = 0; i < iters+1; i++)
    {
        setArrays((cf_t *)A, B, C, &alpha, &beta, M, N, K);
        memcpyPinned(h_A, h_B, h_C, &d_alpha, &d_beta, A, B, C, &alpha, &beta, M, N, K);
        auto mem_h2d_result = Arrays2DeviceWithStreams(d_A, d_B, d_C, h_A, h_B, h_C, M, N, K, divisions, cudaStreamHandles);
        hipEventRecord(start);
        for (int k = 0; k < divisions; k++)
        {
            auto start_point = k*n;
            auto end_point = start_point + n;
            auto _n = end_point < N ? n : N-start_point;
            hipblasCgemm(
                cudaStreamHandles[k].first,
                HIPBLAS_OP_N,
                HIPBLAS_OP_N,
                _n,
                M,
                K,
                &d_alpha,
                &d_B[start_point],
                _n,
                d_A,
                K,
                &d_beta,
                &d_C[start_point],
                _n
            );
        }
        hipEventRecord(stop);

        auto mem_d2h_result = Array2Host(h_C, d_C, M, N, K);

        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);
        
        // 記録
        if (i != 0)
        {
            ms_results.push_back(milliseconds);
            memcpy_h2d_results.push_back(mem_h2d_result.second);
            memcpy_d2h_results.push_back(mem_d2h_result.second);
        }
    }
    printResults(ms_results, memcpy_h2d_results, memcpy_d2h_results);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipHostFree(h_A);
    hipHostFree(h_B);
    hipHostFree(h_C);
    free(A);
    free(B);
    free(C);
    for (int k = 0; k < divisions; k++)
    {
        hipblasDestroy(cudaStreamHandles[k].first);
        hipStreamDestroy(cudaStreamHandles[k].second);
    }
    cudaErrorHandle(hipDeviceReset());
    return 0;
}