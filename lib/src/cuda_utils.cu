#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <iostream>
#include "cuda_utils.cuh"

int cudaErrorHandle(hipError_t result)
{
    if (result == hipSuccess)
    {
        return 0;
    }
    else if (result == hipErrorInvalidValue)
    {
        std::cout << "Error: Invalid value\n";
        return 1;
    }
    else if (result == hipErrorInvalidMemcpyDirection)
    {
        std::cout << "Error: Invalid memory copy direction\n";
        return 1;
    }
    return 2;
}

int memcpyPinned(hipComplex *h_A, hipComplex *h_B, hipComplex *h_C, hipComplex *h_alpha, hipComplex *h_beta, const cf_t *A,
                 const cf_t *B, const cf_t *C, const cf_t *alpha, const cf_t *beta, const int M, const int N,
                 const int K)
{
    if (sizeof(cf_t) != sizeof(hipComplex))
    {
        return 1;
    }
    memcpy(h_A, A, sizeof(cf_t) * M * K);
    memcpy(h_B, B, sizeof(cf_t) * K * N);
    memcpy(h_C, C, sizeof(cf_t) * M * N);
    h_alpha->x = alpha->r;
    h_alpha->y = alpha->i;
    h_beta->x = beta->r;
    h_beta->y = beta->i;
    return 0;
}

std::pair<int, float> Arrays2Device(hipComplex *d_A, hipComplex *d_B, hipComplex *d_C, hipComplex *h_A, hipComplex *h_B,
                                    hipComplex *h_C, int M, int N, int K)
{
    int check = 0;
    hipError_t result;

    // 初期化
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    result = hipMemcpy(d_A, h_A, sizeof(hipComplex) * M * K, hipMemcpyHostToDevice);
    check += cudaErrorHandle(result);
    hipEventRecord(start);
    result = hipMemcpy(d_B, h_B, sizeof(hipComplex) * K * N, hipMemcpyHostToDevice);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    check += cudaErrorHandle(result);
    result = hipMemcpy(d_C, h_C, sizeof(hipComplex) * M * N, hipMemcpyHostToDevice);
    check += cudaErrorHandle(result);
    return std::make_pair(check, milliseconds);
}

std::pair<int, float> Arrays2DeviceWithStreams(
    hipComplex *d_A, hipComplex *d_B, hipComplex *d_C,
    hipComplex *h_A, hipComplex *h_B, hipComplex *h_C,
    int M, int N, int K,
    int divisions, std::vector<cudaStreamHandle_t> &streamHandles)
{
    int check = 0;
    hipError_t result;

    // 初期化
    std::vector<hipEvent_t> starts(divisions), stops(divisions);
    for (int k = 0; k < divisions; k++)
    {
        hipEventCreate(&starts[k]);
        hipEventCreate(&stops[k]);
    }

    // パラメタ設定
    int n = ceil((double)N/divisions);

    // 時間測定結果保管用
    float milliseconds = 0;

    // 非測定部分の行列を先に転送
    result = hipMemcpy(d_A, h_A, sizeof(hipComplex) * M * K, hipMemcpyHostToDevice);
    check += cudaErrorHandle(result);
    result = hipMemcpy(d_C, h_C, sizeof(hipComplex) * M * N, hipMemcpyHostToDevice);
    check += cudaErrorHandle(result);

    
    for (int k = 0; k < divisions; k++)
    {
        // update _n: transfering size
        auto start_point = k*n;
        auto end_point = (k+1)*n;
        auto _n = end_point < N ? n : N-start_point;

        hipEventRecord(starts[k]);
        result = hipMemcpyAsync(
            &d_B[start_point],
            &h_B[start_point],
            sizeof(hipComplex) * K * _n,
            hipMemcpyHostToDevice,
            streamHandles[k].second);
        check += cudaErrorHandle(result);
    }
    for (int k = 0; k < divisions; k++)
    {
        float tmp_milliseconds = 0;
        hipStreamSynchronize(streamHandles[k].second);
        hipEventRecord(stops[k]);
        hipEventSynchronize(stops[k]);
        hipEventElapsedTime(&tmp_milliseconds, starts[k], stops[k]);
        milliseconds += tmp_milliseconds;
    }
    
    
    return std::make_pair(check, milliseconds/divisions);
}

std::pair<int, float> Array2Host(
    hipComplex *h_C, hipComplex *d_C,
    int M, int N, int K)
{
    int check = 0;
    hipError_t result;

    // 初期化
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    result = hipMemcpy(h_C, d_C, sizeof(hipComplex) * M * N, hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    check += cudaErrorHandle(result);
    return std::make_pair(check, milliseconds);
}

std::pair<int, float> Arrays2HostWithStreams(
    hipComplex *d_C, hipComplex *h_C,
    int M, int N, int K,
    int divisions, std::vector<cudaStreamHandle_t> &streamHandles)
{
    int check = 0;
    hipError_t result;

    // 初期化
    std::vector<hipEvent_t> starts(divisions), stops(divisions);
    for (int k = 0; k < divisions; k++)
    {
        hipEventCreate(&starts[k]);
        hipEventCreate(&stops[k]);
    }

    // パラメタ設定
    int n = ceil((double)N/divisions);

    // 時間測定結果保管用
    float milliseconds = 0;

    for (int k = 0; k < divisions; k++)
    {
        // update _n: transfering size
        auto start_point = k*n;
        auto end_point = (k+1)*n;
        auto _n = end_point < N ? n : N-start_point;
        
        hipEventRecord(starts[k]);
        result = hipMemcpyAsync(
            &h_C[start_point],
            &d_C[start_point],
            sizeof(hipComplex) * K * _n,
            hipMemcpyDeviceToHost,
            streamHandles[k].second);
        check += cudaErrorHandle(result);
    }
    for (int k = 0; k < divisions; k++)
    {
        float tmp_milliseconds = 0;
        hipStreamSynchronize(streamHandles[k].second);
        hipEventRecord(stops[k]);
        hipEventSynchronize(stops[k]);
        hipEventElapsedTime(&tmp_milliseconds, starts[k], stops[k]);
        milliseconds += tmp_milliseconds;
    }
    
    return std::make_pair(check, milliseconds/divisions);
}