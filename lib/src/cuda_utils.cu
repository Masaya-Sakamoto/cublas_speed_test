#include <hipblas.h>
#include <iostream>
#include "cuda_utils.cuh"

int cudaErrorHandle(hipError_t result)
{
    if (result == hipSuccess)
    {
        return 0;
    }
    else if (result == hipErrorInvalidValue)
    {
        std::cout << "Error: Invalid value\n";
        return 1;
    }
    else if (result == hipErrorInvalidMemcpyDirection)
    {
        std::cout << "Error: Invalid memory copy direction\n";
        return 1;
    }
    return 2;
}

int memcpyPinned(hipComplex *h_A, hipComplex *h_B, hipComplex *h_C, hipComplex *h_alpha, hipComplex *h_beta, const cf_t *A,
                 const cf_t *B, const cf_t *C, const cf_t *alpha, const cf_t *beta, const int M, const int N,
                 const int K)
{
    if (sizeof(cf_t) != sizeof(hipComplex))
    {
        return 1;
    }
    memcpy(h_A, A, sizeof(cf_t) * M * K);
    memcpy(h_B, B, sizeof(cf_t) * K * N);
    memcpy(h_C, C, sizeof(cf_t) * M * N);
    h_alpha->x = alpha->r;
    h_alpha->y = alpha->i;
    h_beta->x = beta->r;
    h_beta->y = beta->i;
    return 0;
}

std::pair<int, float> Arrays2Device(hipComplex *d_A, hipComplex *d_B, hipComplex *d_C, hipComplex *h_A, hipComplex *h_B,
                                    hipComplex *h_C, int M, int N, int K)
{
    int check = 0;
    hipError_t result;

    // 初期化
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    result = hipMemcpy(d_A, h_A, sizeof(hipComplex) * M * K, hipMemcpyHostToDevice);
    check += cudaErrorHandle(result);
    hipEventRecord(start);
    result = hipMemcpy(d_B, h_B, sizeof(hipComplex) * K * N, hipMemcpyHostToDevice);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    check += cudaErrorHandle(result);
    result = hipMemcpy(d_C, h_C, sizeof(hipComplex) * M * N, hipMemcpyHostToDevice);
    check += cudaErrorHandle(result);
    return std::make_pair(check, milliseconds);
}

std::pair<int, float> Array2Host(hipComplex *h_A, hipComplex *h_B, hipComplex *h_C, hipComplex *d_A, hipComplex *d_B,
                                 hipComplex *d_C, int M, int N, int K)
{
    int check = 0;
    hipError_t result;

    // 初期化
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    result = hipMemcpy(h_A, d_A, sizeof(hipComplex) * M * K, hipMemcpyDeviceToHost);
    check += cudaErrorHandle(result);
    result = hipMemcpy(h_B, d_B, sizeof(hipComplex) * K * N, hipMemcpyDeviceToHost);
    check += cudaErrorHandle(result);
    hipEventRecord(start);
    result = hipMemcpy(h_C, d_C, sizeof(hipComplex) * M * N, hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    check += cudaErrorHandle(result);
    return std::make_pair(check, milliseconds);
}